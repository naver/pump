#include "hip/hip_runtime.h"
// Copyright 2022-present NAVER Corp.
// CC BY-NC-SA 4.0
// Available only for non-commercial use

#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define MIN(x, y)           ((x) < (y) ? (x) : (y))
#define MAX(x, y)           ((x) < (y) ? (y) : (x))
#define inf std::numeric_limits<float>::infinity()

#define CHECK_CUDA(tensor) {\
    TORCH_CHECK((tensor).is_cuda(), #tensor " is not in cuda memory"); \
    TORCH_CHECK((tensor).is_contiguous(), #tensor " is not contiguous"); }
void CHECK_KERNEL() {auto error = hipGetLastError(); TORCH_CHECK( error == hipSuccess, hipGetErrorString(error));}


#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ < 600
#define atomicMax_block atomicMax
#endif


template <typename scalar_t>
__global__ void forward_agg_cuda_kernel( 
        const int LH1, const int LW1, const int LH2, const int LW2, 
        const int gap_left, const int gap_right, float norm,
        const torch::PackedTensorAccessor64<scalar_t,4,torch::RestrictPtrTraits> lower,
              torch::PackedTensorAccessor64<scalar_t,4,torch::RestrictPtrTraits> upper,
        const float* weights, float* new_weights ) {

    const auto UH1 = LH1 + bool(!gap_left); // level 0 is smaller than other levels
    const auto UW1 = LW1 + bool(!gap_left);
    const auto UH2 = LH2;
    const auto UW2 = LW2;

    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int uw2 = idx % UW2; idx /= UW2;
    const int uh2 = idx % UH2; idx /= UH2;
    const int uw1 = idx % UW1; idx /= UW1;
    const int uh1 = idx;
    if (uh1 >= UH1) return;

    // then, add the 4 child
    float sumw = 0, nrm = 0, res = 0;
    // #pragma unroll
    for (int i = 0; i < 4; i++) {
        const int v = i/2, u = i%2;
        // source pixel
        const int lh1 = uh1 + (1-v) * gap_left - v * gap_right;
        if (lh1 < 0 || lh1 >= LH1) continue;
        const int lw1 = uw1 + (1-u) * gap_left - u * gap_right;
        if (lw1 < 0 || lw1 >= LW1) continue;

        // load weight even if (lh2,lw2) are invalid
        const float weight = weights ? weights[lh1*LW1 + lw1] : 1;
        sumw += weight;

        const int lh2 = uh2 + 1 - 2*v;
        if (lh2 < 0 || lh2 >= LH2) continue;
        const int lw2 = uw2 + 1 - 2*u;
        if (lw2 < 0 || lw2 >= LW2) continue;

        res += weight * lower[lh1][lw1][lh2][lw2];
        nrm += weight;
    }

    // normalize output
    nrm = sumw * (nrm < sumw ? powf(nrm/sumw, norm) : 1);
    upper[uh1][uw1][uh2][uw2] = (nrm ? res / nrm : 0);
    if (uh2 == 1 && uw2 == 1)
        new_weights[uh1*UW1 + uw1] = sumw;
}

torch::Tensor forward_agg_cuda( int level, float norm, const torch::Tensor lower, 
                                const at::optional<at::Tensor> weights, torch::Tensor upper ) {
    CHECK_CUDA(lower);
    CHECK_CUDA(upper);
    if (weights) CHECK_CUDA(weights.value());

    const auto UH1 = upper.size(0);
    const auto UW1 = upper.size(1);
    const auto UH2 = upper.size(2);
    const auto UW2 = upper.size(3);
    const auto LH1 = lower.size(0);
    const auto LW1 = lower.size(1);
    const auto LH2 = lower.size(2);
    const auto LW2 = lower.size(3);
    TORCH_CHECK( UH1 == LH1 + int(level==1) && UW1 == LW1 + int(level==1), "inconsistent lower and upper shapes" );

    const int gap_left = (level >= 2) ? 1 << (level-2) : 0; // 0, 1, 2, 4, ...
    const int gap_right= 1 << MAX(0, level-2);              // 1, 1, 2, 4, ...

    const int MAX_THREADS = 512; // faster than 1024 (higher SM occupancy)
    const int THREADS_PER_BLOCK = MAX_THREADS;
    const int N_BLOCKS = (UH1*UW1*UH2*UW2 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    torch::Tensor new_weights = torch::zeros({UH1, UW1}, upper.options().dtype(torch::kFloat32));

    // one block for each layer, one thread per local-max
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(lower.type(), "forward_agg_cuda", ([&] {
        forward_agg_cuda_kernel<<<N_BLOCKS, THREADS_PER_BLOCK>>>(
            LH1, LW1, LH2, LW2, 
            gap_left, gap_right, norm,
            lower.packed_accessor64<scalar_t,4,torch::RestrictPtrTraits>(),
            upper.packed_accessor64<scalar_t,4,torch::RestrictPtrTraits>(),
            weights ? weights->data_ptr<float>() : nullptr, new_weights.data_ptr<float>() );
    }));
    return new_weights;
}

template <typename scalar_t>
__global__ void forward_pool_agg_cuda_kernel( 
        const int LH1, const int LW1, const int LH2, const int LW2, 
        // const int UH1, const int UW1, const int UH2, const int UW2,
        const int gap_left, const int gap_right, float norm,
        const torch::PackedTensorAccessor64<scalar_t,4,torch::RestrictPtrTraits> lower,
              torch::PackedTensorAccessor64<scalar_t,4,torch::RestrictPtrTraits> upper,
        const float* weights, float* new_weights ) {

    const auto UH1 = LH1 + bool(!gap_left); // level 0 is smaller than other levels
    const auto UW1 = LW1 + bool(!gap_left);
    const auto UH2 = (LH2-1)/2 + 1;
    const auto UW2 = (LW2-1)/2 + 1;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int uw2 = idx % UW2; idx /= UW2;
    const int uh2 = idx % UH2; idx /= UH2;
    const int uw1 = idx % UW1; idx /= UW1;
    const int uh1 = idx;
    if (uh1 >= UH1) return;

    // then, add the 4 child
    float sumw = 0, nrm = 0, res = 0;
    // #pragma unroll
    for (int i = 0; i < 4; i++) {
        const int v = i/2, u = i%2;
        // source pixel
        const int lh1 = uh1 + (1-v) * gap_left - v * gap_right;
        if (lh1 < 0 || lh1 >= LH1) continue;
        const int lw1 = uw1 + (1-u) * gap_left - u * gap_right;
        if (lw1 < 0 || lw1 >= LW1) continue;

        // load weight even if (lh2,lw2) are invalid
        const float weight = weights ? weights[lh1*LW1 + lw1] : 1;
        sumw += weight;

        const int lh2_ = 2*(uh2 + 1 - 2*v); // position in lower
        const int lw2_ = 2*(uw2 + 1 - 2*u);
        float lower_max = -inf;
        #pragma unroll
        for (int j = -1; j <= 1; j++) {
          const int lh2 = lh2_ + j;
          if (lh2 < 0 || lh2 >= LH2) continue;
          #pragma unroll
          for (int i = -1; i <= 1; i++) {
            const int lw2 = lw2_ + i;
            if (lw2 < 0 || lw2 >= LW2) continue;
            float l = lower[lh1][lw1][lh2][lw2];
            lower_max = MAX(lower_max, l);
        }}
        if (lower_max == -inf) continue;

        res += weight * lower_max;
        nrm += weight;
    }

    // normalize output
    nrm = sumw * (nrm < sumw ? powf(nrm/sumw, norm) : 1);
    upper[uh1][uw1][uh2][uw2] = (nrm ? res / nrm : 0);
    if (uh2 == 1 && uw2 == 1)
        new_weights[uh1*UW1 + uw1] = sumw;
}

torch::Tensor forward_pool_agg_cuda( int level, float norm, const torch::Tensor lower, 
                                     const at::optional<at::Tensor> weights, torch::Tensor upper ) {
    CHECK_CUDA(lower);
    CHECK_CUDA(upper);
    if (weights) CHECK_CUDA(weights.value());

    const auto LH1 = lower.size(0);
    const auto LW1 = lower.size(1);
    const auto LH2 = lower.size(2);
    const auto LW2 = lower.size(3);
    const auto UH1 = upper.size(0);
    const auto UW1 = upper.size(1);
    const auto UH2 = upper.size(2);
    const auto UW2 = upper.size(3);
    TORCH_CHECK( UH1 == LH1 + int(level==1) && UW1 == LW1 + int(level==1), "inconsistent lower and upper shapes" );
    TORCH_CHECK( UH2 == (LH2-1)/2+1 && UW2 == (LW2-1)/2+1, "lower level should be twice as big" );

    const int gap_left = (level >= 2) ? 1 << (level-2) : 0; // 0, 1, 2, 4, ...
    const int gap_right= 1 << MAX(0, level-2);              // 1, 1, 2, 4, ...

    const int MAX_THREADS = 512; // faster than 1024 (higher SM occupancy)
    const int THREADS_PER_BLOCK = MAX_THREADS;
    const int N_BLOCKS = (UH1*UW1*UH2*UW2 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    torch::Tensor new_weights = torch::zeros({UH1, UW1}, upper.options().dtype(torch::kFloat));
    
    // one block for each layer, one thread per local-max
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(lower.type(), "forward_pool_agg_cuda", ([&] {
        forward_pool_agg_cuda_kernel<<<N_BLOCKS, THREADS_PER_BLOCK>>>(
            LH1, LW1, LH2, LW2, 
            // UH1, UW1, UH2, UW2, 
            gap_left, gap_right, norm,
            lower.packed_accessor64<scalar_t,4,torch::RestrictPtrTraits>(),
            upper.packed_accessor64<scalar_t,4,torch::RestrictPtrTraits>(),
            weights ? weights->data<float>() : nullptr, new_weights.data<float>() );
    }));
    return new_weights;
}

__device__ inline int in(int lower, int var, int upper) {
    return lower <= var && var < upper;
}
__device__ inline int sl(bool b) {
    return b ? 1 : -1;
}

__device__ short atomicMaxShort(short* address, short val) {
    unsigned int *base_address = (unsigned int *)((size_t)address & ~3); // multiple of 4
    
    unsigned int order_from[] = {0x0010, 0x0032}; // either bytes[0:2] or bytes[2:4]
    unsigned int from = order_from[((size_t)address & 3) / 2];
    
    unsigned int order_back[] = {0x3254, 0x5410}; // right-to-left 
    unsigned int back = order_back[((size_t)address & 3) / 2];
    unsigned int old, assumed, max_, new_;

    old = *base_address;
    do {
        assumed = old;
        max_ = max(val, (short)__byte_perm(old, 0, from)); // extract word
        new_ = __byte_perm(old, max_, back); // replace word
        old = atomicCAS(base_address, assumed, new_);
    } while (assumed != old);
    return old;
}

template <typename scalar_t>
__device__ inline void TplAtomicMax_block( scalar_t* before, scalar_t after ) { assert(!"atomicMax not implemented for this dtype"); }
template <>
__device__ inline void TplAtomicMax_block( at::Half* before, at::Half after ) { atomicMaxShort( (int16_t*)before, *(int16_t*)&after ); }
template <>
__device__ inline void TplAtomicMax_block( float* before, float after ) { atomicMax_block( (int32_t*)before, *(int32_t*)&after ); }

template <typename scalar_t>
__global__ void backward_agg_unpool_cuda_kernel( 
        const int UH1, const int UW1, 
        const int UH2, const int UW2, 
        const int LH2, const int LW2, 
        const int gap_left, const int gap_right,
        const torch::PackedTensorAccessor64<scalar_t,4,torch::RestrictPtrTraits> upper,
              torch::PackedTensorAccessor64<scalar_t,4,torch::RestrictPtrTraits> lower ) {

    /* Each block is going to take care of a single layer, i.e. lower[:,:,0::2,0::2].
       the first thread is allocating some global memory and then frees it later.
    */
    // const int LH1 = gridDim.x;
    // const int LW1 = gridDim.y;
    const int lh1 = blockIdx.y;
    const int lw1 = blockIdx.x;
    const int UHW2 = UH2 * UW2; // upper layer size

    __shared__ float* _shared_addr;
    if (threadIdx.x == 0)
        do{ _shared_addr = new float [2*UHW2]; } // for each upper place, we have (best, bestp)
        while(!_shared_addr); // waiting for memory to be available...
    __syncthreads();

    float * layer_best = _shared_addr;
    int * layer_bestp = (int*)(_shared_addr+1); //UHW);
    assert( layer_best );

    /* First pass: we recover the position and values of all local maxima in the layer
    */ 
    for (int idx = threadIdx.x; idx < UHW2; idx += blockDim.x) {
        const int ux = idx % UW2;
        const int uy = idx / UW2;
        const int lx = 2*ux; // lower pos from upper pos
        const int ly = 2*uy;

        // argmax my local minima
        float best = -inf;
        int bestp = 0;
        #pragma unroll
        for (int j_= -1; j_<= 1; j_++) {
          const int j = ly + j_;
          if (j < 0 || j >= LH2) continue;
          #pragma unroll
          for (int i_= -1; i_<= 1; i_++) {
            const int i = lx + i_;
            if (i < 0 || i >= LW2) continue;
            float cur = lower[lh1][lw1][j][i];
            if (cur > best) { best = cur; bestp = j*LW2+i; }
        }}
        layer_best[2*idx] = best;
        layer_bestp[2*idx] = bestp;
    }
    
    __syncthreads();
    
    /* Second pass: we update the local maxima according to the upper layer
    */ 
    for (int idx = threadIdx.x; idx < UHW2; idx += blockDim.x) {
        const int ux = idx % UW2;
        const int uy = idx / UW2;

        // max-pool the additional value from the upper layer
        scalar_t add = 0;
        for (int v = -gap_left; v <= gap_right; v += gap_right+gap_left) {
          for (int u = -gap_left; u <= gap_right; u += gap_right+gap_left) {
            const int uh1 = lh1 + v, uw1 = lw1 + u;
            const int uh2 = uy+sl(v>0), uw2 = ux+sl(u>0);
            if (in(0, uh1, UH1) && in(0, uw1, UW1) && in(0, uh2, UH2) && in(0, uw2, UW2))
                add = MAX(add, upper[uh1][uw1][uh2][uw2]);
        }}

        // grab local maxima
        float best = layer_best[2*idx];
        int bestp = layer_bestp[2*idx];
        const int lx = bestp % LW2;
        const int ly = bestp / LW2;

        // printf("UH=%d,UW=%d: uy=%d,ux=%d --> best=%g at ly=%d,lx=%d\n", UH,UW, uy,ux, best, ly,lx);
        scalar_t* before = & lower[lh1][lw1][ly][lx];
        scalar_t  after  = best + add;
        TplAtomicMax_block<scalar_t>( before, after );
    }

    __syncthreads();

    if (threadIdx.x == 0) 
        delete _shared_addr;
}

void backward_agg_unpool_cuda( int level, const torch::Tensor upper, torch::Tensor lower, bool exclude_borders ) {
    CHECK_CUDA(lower);
    CHECK_CUDA(upper);

    const auto UH1 = upper.size(0);
    const auto UW1 = upper.size(1);
    const auto UH2 = upper.size(2);
    const auto UW2 = upper.size(3);
    const auto LH1 = lower.size(0);
    const auto LW1 = lower.size(1);
    const auto LH2 = lower.size(2);
    const auto LW2 = lower.size(3);
    TORCH_CHECK( UH1 == LH1 + int(level==1) && UW1 == LW1 + int(level==1), "inconsistent lower and upper shapes" );
    const int xb = exclude_borders; // local_argmax cannot reach the bottom and right borders

    const int gap_left = (level >= 2) ? 1 << (level-2) : 0; // 0, 1, 2, 4, ...
    const int gap_right= 1 << MAX(0, level-2);              // 1, 1, 2, 4, ...

    const int64_t MAX_THREADS = 1024;
    const int64_t THREADS_PER_LAYER = MIN(UH2*UW2, MAX_THREADS);

    // one block for each layer, one thread per local-max
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(upper.type(), "backward_agg_unpool_cuda", ([&] {
        backward_agg_unpool_cuda_kernel<<<dim3(LW1,LH1), THREADS_PER_LAYER>>>(
            UH1, UW1, UH2, UW2, LH2-xb, LW2-xb, 
            gap_left, gap_right,
            upper.packed_accessor64<scalar_t,4,torch::RestrictPtrTraits>(),
            lower.packed_accessor64<scalar_t,4,torch::RestrictPtrTraits>());
    }));
    CHECK_KERNEL();
}

template <typename scalar_t>
__global__ void max_pool3d_cuda_kernel( 
        const int BS, const int NC, const int IH, const int IW, const int OH, const int OW, 
        const int ks, const int stride,
        const torch::PackedTensorAccessor64<scalar_t,4,torch::RestrictPtrTraits> tensor,
              torch::PackedTensorAccessor64<scalar_t,3,torch::RestrictPtrTraits> maxima,
              torch::PackedTensorAccessor64<int64_t,    3,torch::RestrictPtrTraits> indices ) {

    // each thread takes care of one output
    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int x = idx % OW; idx /= OW;
    const int y = idx % OH; idx /= OH;
    const int b = idx;
    if (b >= BS) return;

    float best = -inf;
    int64_t best_pos = 0;
    for (int64_t c = 0; c < NC; c++) {
      for (int j = stride*y; j < stride*y+ks; j++) {
        for (int i = stride*x; i < stride*x+ks; i++) {
            // assert( b < BS and c < NC and j < IH and i < IW );
            float cur = tensor[b][c][j][i];
            if (cur > best) {best = cur; best_pos = (c*IH + j)*IW+ i; }
    }}}

    // assert( b < BS and y < OH and x < OW );
    maxima [b][y][x] = best;
    indices[b][y][x] = best_pos;
}

void max_pool3d_cuda( const torch::Tensor tensor, const int kernel_size, const int stride,
                            torch::Tensor maxima, torch::Tensor indices ) {
    CHECK_CUDA(tensor);
    TORCH_CHECK(tensor.dim() == 4, "tensor should be 4-dimensional: BxCxHxW");
    const int BS = tensor.size(0);
    const int NC = tensor.size(1);
    const int IH = tensor.size(2); // input height
    const int IW = tensor.size(3); // input width

    // output size
    TORCH_CHECK( maxima.sizes() == indices.sizes(), "maxima and indices should have the same shape" );
    TORCH_CHECK( BS == maxima.size(0), "bad batch size" );
    const int OH = maxima.size(1);
    const int OW = maxima.size(2);

    const int64_t THREADS_PER_LAYER = 512;
    const int64_t N_BLOCKS = (BS*OH*OW + THREADS_PER_LAYER-1) / THREADS_PER_LAYER;
    
    // one block for each layer, one thread per local-max
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(tensor.type(), "max_pool3d_cuda", ([&] {
       max_pool3d_cuda_kernel<<<N_BLOCKS, THREADS_PER_LAYER>>>(
            BS, NC, IH, IW, OH, OW, kernel_size, stride,
            tensor. packed_accessor64<scalar_t,4,torch::RestrictPtrTraits>(),
            maxima. packed_accessor64<scalar_t,3,torch::RestrictPtrTraits>(),
            indices.packed_accessor64<int64_t,3,torch::RestrictPtrTraits>());
    }));
}


__device__ inline float ptdot( const float* m, float x, float y ) {
  return x*m[0] + y*m[1] + m[2];
}

__device__ inline float sqr(float v) {
    return v*v;
}


__global__ void merge_corres_cuda_kernel( 
            const int OH, const int OW, const int OZ, const int IH, const int IW, 
            const float dmax2, int offset, const float* inv_rot, const int all_step,
            const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> corres_a,
                  torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> all_corres_a ) {

    // each thread takes care of one output
    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int i = idx % OW; idx /= OW;
    const int j = idx;
    if (j >= OH) return;

    const float tol2 = 2*2; // squared
    auto all_cor = all_corres_a[j][i];
    
    // center of the bin in the reference frame
    float x = i*all_step + all_step/2;
    float y = j*all_step + all_step/2;

    // center of the bin on the rescaled+rotated image
    float xr = ptdot( inv_rot + 0, x, y ); 
    float yr = ptdot( inv_rot + 3, x, y );

    // iterate on the nearby bins
    int xb = (int)(0.5+ xr/4); // rescaled+rotated desc always has step 4
    int yb = (int)(0.5+ yr/4);
    
    float best = dmax2;
    #pragma unroll
    for (int _v = -1; _v <= 1; _v++) {
      #pragma unroll
      for (int _u = -1; _u <= 1; _u++) {
        const int v = yb+_v, u = xb+_u;
        if (!(in(0, v, IH) && in(0, u, IW))) continue;
        auto cor = corres_a[v][u];
        float d = sqr(cor[offset]-x) + sqr(cor[offset+1]-y);
        if (d < best)  best = d;
    }}

    #pragma unroll
    for (int _v = -1; _v <= 1; _v++) {
      #pragma unroll
      for (int _u = -1; _u <= 1; _u++) {
        const int v = yb+_v, u = xb+_u;
        if (!(in(0, v, IH) && in(0, u, IW))) continue;
        auto cor = corres_a[v][u];
        float d = sqr(cor[offset]-x) + sqr(cor[offset+1]-y);
        if (d <= tol2*best) { // spatially close
            // merge correspondence if score is better than actual
            if (cor[4] > all_cor[4])
              for (int k = 0; k < OZ; k++) all_cor[k] = cor[k];
          }
    }}
}

void merge_corres_cuda( const torch::Tensor corres, const int offset, const torch::Tensor _inv_rot, 
                        const float dmax, torch::Tensor all_corres, const int all_step ) {
    CHECK_CUDA( corres );
    CHECK_CUDA( all_corres );
    CHECK_CUDA( _inv_rot );
    TORCH_CHECK(_inv_rot.is_contiguous(), "inv_rot should be contiguous" );

    const int IH = corres.size(0);
    const int IW = corres.size(1);
    const int IZ = corres.size(2);
    const int OH = all_corres.size(0);
    const int OW = all_corres.size(1);
    const int OZ = all_corres.size(2);
    TORCH_CHECK( IZ == OZ, "corres and all_corres should have the same shape[2]" );

    const int THREADS_PER_LAYER = 512;
    const int N_BLOCKS = (OH * OW + THREADS_PER_LAYER-1) / THREADS_PER_LAYER;
    
    merge_corres_cuda_kernel<<<N_BLOCKS, THREADS_PER_LAYER>>>(
        OH, OW, OZ, IH, IW, dmax*dmax, offset, _inv_rot.data_ptr<float>(), all_step,
                corres.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
            all_corres.packed_accessor32<float,3,torch::RestrictPtrTraits>());
    CHECK_KERNEL();
}


template <typename scalar_t>
__global__ void mask_correlations_radial_cuda_kernel( 
            float radius, const float alpha,
            const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> targets,
                  torch::PackedTensorAccessor64<scalar_t,4,torch::RestrictPtrTraits> corr ) {

    #define H1 ((int)corr.size(0))
    #define W1 ((int)corr.size(1))
    #define H2 ((int)corr.size(2))
    #define W2 ((int)corr.size(3))

    // each block takes care of one layer corr[j,i,:,:]
    const int j = blockIdx.x / W1;
    const int i = blockIdx.x % W1;
    if (j >= H1) return;

    // read the target center
    const float cx = targets[j][i][0];
    const float cy = targets[j][i][1];
    if (cx != cx || cy != cy) return; // undefined center
    radius *= radius; // squared
    const float alpha_out = (alpha > 1 ? 1 : alpha);
    const float alpha_in = (alpha < 1 ? 1 : alpha);
    
    for (int idx = threadIdx.x; idx < H2*W2; idx += blockDim.x) {
        const int v = idx / W2;
        const int u = idx % W2;

        // compute weighting
        float dis2 = sqr(u - cx) + sqr(v - cy);
        float mul = alpha_in;
        if (dis2 > radius) 
            mul = 1 - alpha_out*(1 - radius / dis2);

        corr[j][i][v][u] *= mul; 
    }
}

void mask_correlations_radial_cuda( torch::Tensor corr, const torch::Tensor targets, 
                                    const float radius, const float alpha) {
    CHECK_CUDA( corr );
    CHECK_CUDA( targets );

    const int THREADS_PER_LAYER = 512;
    const int N_BLOCKS = H1*W1;

    #undef H1
    #undef W1
    #undef H2
    #undef W2

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(corr.type(), "mask_correlations_radial_cuda", ([&] {    
        mask_correlations_radial_cuda_kernel<<<N_BLOCKS, THREADS_PER_LAYER>>>(
            radius, alpha,
            targets.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
               corr.packed_accessor64<scalar_t,4,torch::RestrictPtrTraits>());
    }));
    CHECK_KERNEL();
}
